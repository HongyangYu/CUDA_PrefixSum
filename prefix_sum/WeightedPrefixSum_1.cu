
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10000003 //10,000,000,  1024 is ok
#define BLOCKSIZE 1024

/* prefix sum */

using namespace std;

__global__ void add(int* in, int* out, int offset, int n){
	
	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	if(gid >= n) return ;
	
	out[gid] = in[gid];
	if(gid >= offset)
		out[gid] += in[gid-offset]; 
}

int main(){
	int *in, *out;
	int *d_in, *d_out;
	
	int size = N * sizeof(int);
	// Allocate space for host of in, out
	in = (int *)malloc(size);
	out = (int *)malloc(size);
	
	// Allocate space for device copies of in, out
	hipMalloc((void **)&d_in, size);
	hipMalloc((void **)&d_out, size);
	
	//initialize input;
	for(int i=0; i<N; i++){
		in[i] = 1;
		out[i] = 0;
	}
	
	// Copy inputs to device
	hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	int isOdd = 1;
	//prefix sum once
	for(int offset=1; offset<N; offset = (offset<<1)){
		if(isOdd == 1){ //odd
			add<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_in, d_out, offset, N);
			isOdd = 0;
		} else { //even
			add<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_out, d_in, offset, N);
			isOdd = 1;
		}
	}
	//prefix sum twice
	for(int offset=1; offset<N; offset = (offset<<1)){
		if(isOdd == 1){ //odd
			add<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_in, d_out, offset, N);
			isOdd = 0;
		} else { //even
			add<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_out, d_in, offset, N);
			isOdd = 1;
		}
	}
	
	//Copy memory from GPU to CPU 
	hipMemcpy(in, d_in, size, hipMemcpyDeviceToHost);
	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
			
	for(int i=0; i<N; i++){
		if(isOdd==0){
			if(out[i]!=(i+1)*(i+2)/2) 
				cout<< i << " : " << out[i] <<endl;
		} else {
			if(in[i]!=(i+1)*(i+2)/2) 
				cout<< i << " : " << in[i] <<endl;
		}
		
	}
	
	cout<< "+=============+"<<endl;
	cout<< "N = " << N <<endl;
	if(isOdd==0){
		cout<< (N-1) << " : " << out[N-1] <<endl;
	} else {
		cout<< (N-1) << " - " << in[N-1] <<endl;
	}

	//Clean Host
	free(in); free(out);
	
	//Clean Device
	hipFree(d_in);
	hipFree(d_out);
	
	return 0;
}