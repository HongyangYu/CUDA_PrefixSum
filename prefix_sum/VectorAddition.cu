#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
//#include "hip/hip_runtime.h"
//#include ""

#define THREAD 1024
#define POWER 25 // =  log(n / THREAD) 

using namespace std;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x; 
	if (i < n) {
		y[i] = x[i] + y[i];
	}
}

int main(void)
{
	int n = 1 << POWER;

	float *x, *y, *d_x, *d_y;

	//Allocate memory on CPU
	x = (float*)malloc(n * sizeof(float));
	y = (float*)malloc(n * sizeof(float));

	//Allocate memory on GPU
	hipMalloc(&d_x, n * sizeof(float));
	hipMalloc(&d_y, n * sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < n; i++) {
		x[i] = 3.0f;
		y[i] = 2.0f;
	}

	//Copy memory from CPU to GPU 
	hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

        //Sequential add vectors
        clock_t begin = clock();
        for (int i = 0; i < n; i++) {
               y[i] = x[i] + y[i];
        }
        clock_t end = clock();
        double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
        cout<<"The running time for sequential addtition is " << time_spent << " miliseconds."<<endl;


        begin = clock();
        // Perform Addition on GPU

        add << <(n + THREAD - 1) / THREAD, THREAD >> >(n, d_x, d_y); // use (n + THREAD - 1) / THREAD to get ceiling
        end = clock();
        time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
        cout<<"The running time for parallel addition is ";
        cout<< time_spent << " miliseconds."<<endl;


	//Copy memory from GPU to CPU 
	hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);


	bool flag = true;
	//Verify Results
	for (int i = 0; i < n; i++){
		if (y[i] != 5.0) {
			cout<<"Incorrect Result"<<endl;
			flag = false;
			break;
		}
	}
	if (flag) {
		cout<<"Correct! Welcome to CUDA world!"<<endl;
		count<<"This is message from Henry."<<endl;
	}
	//Free memory on GPU
	hipFree(d_x);
	hipFree(d_y);

	//Free memory on CPU
	free(x);
	free(y);

	return 0;
}