
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10000003 //10,000,000  1024 is ok
#define BLOCKSIZE 1024

/* prefix sum */

using namespace std;

void add_seq(double* in_seq, double* out_seq) {
	out_seq[0] = in_seq[0];
	for(int i=1; i<N; i++){
		out_seq[i] =  out_seq[i-1] + in_seq[i];
	}
	for(int i=1; i<N; i++){
		out_seq[i] += out_seq[i-1];
	}
}

__global__ void add(double* in, double* out, int offset, int n){
	
	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	if(gid >= n) return ;
	
	out[gid] = in[gid];
	if(gid >= offset)
		out[gid] += in[gid-offset]; 
}

int main(){
	double *in, *out, *in_seq, *out_seq;
	double *d_in, *d_out;
	
	int size = N * sizeof(double);
	// Allocate space for host of in, out
	in = (double *)malloc(size);
	out = (double *)malloc(size);
	in_seq = (double *)malloc(size);
	out_seq = (double *)malloc(size);
	
	// Allocate space for device copies of in, out
	hipMalloc((void **)&d_in, size);
	hipMalloc((void **)&d_out, size);
	
	//initialize input;
	for(int i=0; i<N; i++){
		in[i] = 1.0;
		out[i] = 0.0;
		in_seq[i] = 1.0;
		out_seq[i] = 0.0;
	}
	
	// Copy inputs to device
	hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
	
	cout<<"Parallel Calcualtion Start"<<endl;
	
	clock_t t1 = clock();
	
	// Launch add() kernel on GPU
	int isOdd = 1;
	//prefix sum once
	for(int offset=1; offset<N; offset = (offset<<1)){
		if(isOdd == 1){ //odd
			add<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_in, d_out, offset, N);
			isOdd = 0;
		} else { //even
			add<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_out, d_in, offset, N);
			isOdd = 1;
		}
	}
	//prefix sum twice
	for(int offset=1; offset<N; offset = (offset<<1)){
		if(isOdd == 1){ //odd
			add<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_in, d_out, offset, N);
			isOdd = 0;
		} else { //even
			add<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_out, d_in, offset, N);
			isOdd = 1;
		}
	}
	
	clock_t t2 = clock();
	
	cout<<"Parallel Calcualtion End"<<endl;
	
	double time_spent = (double)(t2 - t1) / CLOCKS_PER_SEC * 1000;
	cout << "The running time of parallel addtition is " << time_spent << " miliseconds." << endl;
	
	//Copy memory from GPU to CPU 
	
	if(isOdd==0){
		hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
	} else {
		hipMemcpy(out, d_in, size, hipMemcpyDeviceToHost);
	}
	

	//check parallel calcuation result
	for(int i=0; i<N; i++){
		if(out[i]!=(i+1L)*(i+2L)/2L) 
			cout<< i << " : " << out[i] <<endl;
	}
	
	clock_t t3 = clock();

	add_seq(in_seq, out_seq);

	clock_t t4 = clock();
	
	cout<<"Sequential Calcualtion End"<<endl;
	
	double time_spent2 = (double)(t4 - t3) / CLOCKS_PER_SEC * 1000;
	cout << "The running time of sequential addtition is " << time_spent2 << " miliseconds." << endl;

	//check sequential calcuation result
	for(int i=0; i<N; i++){
		 if(out_seq[i]!=(i+1L)*(i+2L)/2L)
			cout<<i<<","<<out_seq[i]<<endl;
	}
	
	cout<< "+=============+"<<endl;
	cout<< "N = " << N <<" , N*(N+1)/2 = "<< N*(N+1L)/2L <<endl;
	
	cout<< (N-1) << " : " << (long)out[N-1] << (long)out_seq[N-1] <<endl;
	
	//Clean Device
	hipFree(d_in);
	hipFree(d_out);

	//Clean Host
	free(in); free(out);
	
	
	return 0;
}