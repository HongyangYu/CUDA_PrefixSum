
#include <hip/hip_runtime.h>
#include <iostream>

#define N 32
#define LOG 5
#define BLOCKSIZE 8

/* 一维数Radius相加 share memory*/

using namespace std;

__global__ void add(int* in, int d, int n){
	
	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	if(gid >= n) return ;
	
	int pre = (d==0) ? 1 : (2<<(d-1));
	
	if(gid >= pre) {
		in[gid] += in[gid-pre]; 
	}
}

int main(){
	int *in, *out;
	int *d_a;
	
	int size = N * sizeof(int);
	// Allocate space for host of in, out
	in = (int *)malloc(size);
	out = (int *)malloc(size);
	
	// Allocate space for device copies of in
	hipMalloc((void **)&d_a, size);
	
	
	//initialize input;
	for(int i=0; i<N; i++){
		in[i] = 1;
		out[i] = 0;
	}
	
	
	// Launch add() kernel on GPU
	for(int d=0; d<=LOG; d++){
		// Copy inputs to device
		hipMemcpy(d_a, in, size, hipMemcpyHostToDevice);
		add<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(d_a, d, N);
		hipMemcpy(in, d_a, size, hipMemcpyDeviceToHost);
	}
	
	//Copy memory from GPU to CPU 
	hipMemcpy(out, d_a, size, hipMemcpyDeviceToHost);
			
	for(int i=0; i<N; i++){
		cout<< i << " : " << in[i] << " , " << out[i] <<endl;
	}
	
	//Clean Host
	free(in); free(out);
	
	//Clean Device
	hipFree(d_a);
	
	return 0;
}